#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <time.h>

// replace with larger vector size to test
#define N (1024) 

// add function cuda
__global__ void add(float *a, float *b, float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the thread's unique global index
    c[tid] = a[tid] + b[tid];
}

int main(int argc, char *argv[])
{
    hipSetDevice(0);

    int i;
    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    // allocate memory on the CPU
    a = (float *) malloc(N*sizeof(float));
    b = (float *) malloc(N*sizeof(float));
    c = (float *) malloc(N*sizeof(float));

    // allocate the memory on the GPU
    hipMalloc((void **) &dev_a, N * sizeof(float));
    hipMalloc((void **) &dev_b, N * sizeof(float));
    hipMalloc((void **) &dev_c, N * sizeof(float));

    // fill the arrays 'a' and 'b' on the CPU
    for (i=0; i<N; i++) {
        a[i] = sin(i) * sin(i);
        b[i] = cos(i) * cos(i);
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    clock_t startTime = clock();

    // add the arrays 'a' and 'b' on the GPU
    add<<<N / 1024,1024>>>(dev_a, dev_b, dev_c);

    // Get the ending time
    clock_t endTime = clock();

    // Calculate the elapsed time in seconds
    double deltaTime = (double) (endTime - startTime) / CLOCKS_PER_SEC;

    // Print the delta time
    printf("Vector size %d - Delta time gpu: %lf seconds\n", N, deltaTime);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N*sizeof(float), hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "add kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        // return 1;
    }

    startTime = clock();

    // display the results
    for (i=0; i < N; i++) {
	c[i] = a[i] + b[i];
    }

    endTime = clock();

    deltaTime = (double) (endTime - startTime) / CLOCKS_PER_SEC;

    // Print the delta time
    printf("Vector size %d - Delta time cpu: %lf seconds\n", N, deltaTime);

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // free the memory allocated on the CPU
    free(a);
    free(b);
    free(c);

    return 0;
}



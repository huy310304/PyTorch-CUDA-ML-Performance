#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 256

// CUDA kernel for data initialization
__global__ void initialize_data(float* X, float* y, float weight, float bias, float start, float step, int n, hiprandState* states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hiprand_init(1234, i, 0, &states[i]);
        X[i] = start + i * step;
        y[i] = weight * X[i] + bias + hiprand_normal(&states[i]) * 0.1f; // Add some noise
    }
}

// CUDA kernel for forward pass, loss calculation, and gradient calculation using shared memory
__global__ void fused_kernel(float* X, float* y_true, float* w, float* b, float* y_pred, float* loss, float* w_grad, float* b_grad, int n) {
    __shared__ float s_X[BLOCK_SIZE];
    __shared__ float s_y_true[BLOCK_SIZE];
    __shared__ float s_loss[BLOCK_SIZE];
    __shared__ float s_w_grad[BLOCK_SIZE];
    __shared__ float s_b_grad[BLOCK_SIZE];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    if (i < n) {
        s_X[tid] = X[i];
        s_y_true[tid] = y_true[i];
    }
    __syncthreads();

    float local_loss = 0.0f;
    float local_w_grad = 0.0f;
    float local_b_grad = 0.0f;

    if (i < n) {
        // Forward pass
        float pred = w[0] * s_X[tid] + b[0];
        y_pred[i] = pred;

        // Calculate the difference between prediction and true value
        float diff = pred - s_y_true[tid];

        // Compute local loss and gradients
        local_loss = diff * diff / (2 * n);  // Mean Squared Error
        local_w_grad = diff * s_X[tid] / n;
        local_b_grad = diff / n;
    }

    // Store local results in shared memory
    s_loss[tid] = local_loss;
    s_w_grad[tid] = local_w_grad;
    s_b_grad[tid] = local_b_grad;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_loss[tid] += s_loss[tid + stride];
            s_w_grad[tid] += s_w_grad[tid + stride];
            s_b_grad[tid] += s_b_grad[tid + stride];
        }
        __syncthreads();
    }

    // Write the block results to global memory
    if (tid == 0) {
        atomicAdd(loss, s_loss[0]);
        atomicAdd(w_grad, s_w_grad[0]);
        atomicAdd(b_grad, s_b_grad[0]);
    }
}

// CUDA kernel for updating weights
__global__ void update_weights(float* w, float* w_grad, float* b, float* b_grad, float lr) {
    w[0] -= lr * w_grad[0];
    b[0] -= lr * b_grad[0];
}

int main() {
    hipSetDevice(0);

    hipEvent_t start_time, stop;
    hipEventCreate(&start_time);
    hipEventCreate(&stop);

    // Data parameters
    const float weight = 0.7f;
    const float bias = 0.3f;
    const float start = 0.0f;
    const float end = 10.0f;
    const float step = 0.0000001f;

    const int n = static_cast<int>((end - start) / step);
    const int train_split = static_cast<int>(0.8 * n);
    const int test_split = n - train_split;

    // print out the number of data points
    std::cout << "Total number of datapoints: " << n << std::endl;
    std::cout << "Number of training datapoints: " << train_split << std::endl;
    std::cout << "Number of testing datapoints: " << test_split << std::endl;

    // Create CUDA streams
    const int numberOfStreams = 4;
    hipStream_t streams[numberOfStreams];
    for (int i = 0; i < numberOfStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Allocate memory on the device
    float *d_X, *d_y, *d_w, *d_b, *d_y_pred, *d_loss, *d_w_grad, *d_b_grad;
    hiprandState *d_states;
    hipMalloc(&d_X, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_y_pred, n * sizeof(float));
    hipMalloc(&d_loss, sizeof(float));
    hipMalloc(&d_w, sizeof(float));
    hipMalloc(&d_b, sizeof(float));
    hipMalloc(&d_w_grad, sizeof(float));
    hipMalloc(&d_b_grad, sizeof(float));
    hipMalloc(&d_states, n * sizeof(hiprandState));

    hipEventRecord(start_time);

    // Initialize data on GPU
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    initialize_data<<<blocks, BLOCK_SIZE>>>(d_X, d_y, weight, bias, start, step, n, d_states);

    // Initialize weights
    float h_w = 0.83f;
    float h_b = 0.7645f;
    hipMemcpyAsync(d_w, &h_w, sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_b, &h_b, sizeof(float), hipMemcpyHostToDevice, streams[0]);

    // Training loop
    int epochs = 200;
    float lr = 0.01f;

    for (int epoch = 0; epoch < epochs; ++epoch) {
        // Zero the gradients and loss
        float zero = 0.0f;
        hipMemcpyAsync(d_loss, &zero, sizeof(float), hipMemcpyHostToDevice, streams[0]);
        hipMemcpyAsync(d_w_grad, &zero, sizeof(float), hipMemcpyHostToDevice, streams[1]);
        hipMemcpyAsync(d_b_grad, &zero, sizeof(float), hipMemcpyHostToDevice, streams[2]);

        // Fused kernel for training data (using multiple streams)
        int segmentSize = train_split / numberOfStreams;
        for (int i = 0; i < numberOfStreams; ++i) {
            int offset = i * segmentSize;
            fused_kernel<<<(segmentSize + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, streams[i]>>>(
                &d_X[offset], &d_y[offset], d_w, d_b, 
                &d_y_pred[offset], d_loss, d_w_grad, d_b_grad, segmentSize);
        }

        // Update weights
        update_weights<<<1, 1, 0, streams[0]>>>(d_w, d_w_grad, d_b, d_b_grad, lr);

        // Print out what's happening every 100 epochs
        if (epoch % 10 == 0) {
            float h_loss, h_w, h_b;
            hipMemcpyAsync(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost, streams[0]);
            hipMemcpyAsync(&h_w, d_w, sizeof(float), hipMemcpyDeviceToHost, streams[1]);
            hipMemcpyAsync(&h_b, d_b, sizeof(float), hipMemcpyDeviceToHost, streams[2]);
            hipDeviceSynchronize();
            std::cout << "Epoch: " << epoch << " | Loss: " << h_loss << " | w: " << h_w << " | b: " << h_b << std::endl;
        }
    }

    // Compute test loss
    float h_test_loss;
    hipMemsetAsync(d_loss, 0, sizeof(float), streams[0]);
    fused_kernel<<<(test_split + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, streams[0]>>>(
        &d_X[train_split], &d_y[train_split], d_w, d_b, &d_y_pred[train_split], d_loss, d_w_grad, d_b_grad, test_split);
    hipMemcpyAsync(&h_test_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost, streams[0]);

    // Copy final weights and bias back to host
    float h_w_final, h_b_final;
    hipMemcpyAsync(&h_w_final, d_w, sizeof(float), hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(&h_b_final, d_b, sizeof(float), hipMemcpyDeviceToHost, streams[1]);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_time, stop);

    std::cout << "Final weight: " << h_w_final << " (True: " << weight << ")" << std::endl;
    std::cout << "Final bias: " << h_b_final << " (True: " << bias << ")" << std::endl;
    std::cout << "Test loss: " << h_test_loss << std::endl;
    std::cout << "Training time: " << milliseconds << " ms" << std::endl;

    // Free device memory
    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_y_pred);
    hipFree(d_loss);
    hipFree(d_w);
    hipFree(d_b);
    hipFree(d_w_grad);
    hipFree(d_b_grad);
    hipFree(d_states);

    // Destroy streams
    for (int i = 0; i < numberOfStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    // Destroy events
    hipEventDestroy(start_time);
    hipEventDestroy(stop);

    return 0;
}

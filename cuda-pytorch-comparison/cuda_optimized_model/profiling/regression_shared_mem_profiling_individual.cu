#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 256

// CUDA kernel for data initialization
__global__ void initialize_data(float* X, float* y, float weight, float bias, float start, float step, int n, hiprandState* states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hiprand_init(1234, i, 0, &states[i]);
        X[i] = start + i * step;
        y[i] = weight * X[i] + bias + hiprand_normal(&states[i]) * 0.1f; // Add some noise
    }
}

// CUDA kernel for forward pass
__global__ void forward_pass(float* X, float* w, float* b, float* y_pred, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y_pred[i] = w[0] * X[i] + b[0];
    }
}

// CUDA kernel for loss calculation
__global__ void calculate_loss(float* y_pred, float* y_true, float* loss, int n) {
    __shared__ float s_loss[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    float local_loss = 0.0f;
    if (i < n) {
        float diff = y_pred[i] - y_true[i];
        local_loss = diff * diff / (2 * n);  // Mean Squared Error
    }

    s_loss[tid] = local_loss;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_loss[tid] += s_loss[tid + stride];
        }
        __syncthreads();
    }

    // Write the block result to global memory
    if (tid == 0) {
        atomicAdd(loss, s_loss[0]);
    }
}

// CUDA kernel for gradient calculation
__global__ void calculate_gradients(float* X, float* y_pred, float* y_true, float* w_grad, float* b_grad, int n) {
    __shared__ float s_w_grad[BLOCK_SIZE];
    __shared__ float s_b_grad[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    float local_w_grad = 0.0f;
    float local_b_grad = 0.0f;
    if (i < n) {
        float diff = y_pred[i] - y_true[i];
        local_w_grad = diff * X[i] / n;
        local_b_grad = diff / n;
    }

    s_w_grad[tid] = local_w_grad;
    s_b_grad[tid] = local_b_grad;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_w_grad[tid] += s_w_grad[tid + stride];
            s_b_grad[tid] += s_b_grad[tid + stride];
        }
        __syncthreads();
    }

    // Write the block results to global memory
    if (tid == 0) {
        atomicAdd(w_grad, s_w_grad[0]);
        atomicAdd(b_grad, s_b_grad[0]);
    }
}

// CUDA kernel for updating weights
__global__ void update_weights(float* w, float* w_grad, float* b, float* b_grad, float lr) {
    w[0] -= lr * w_grad[0];
    b[0] -= lr * b_grad[0];
}

int main() {
    hipSetDevice(0);

    // Timing events
    hipEvent_t start_time, stop, epoch_start, epoch_stop;
    hipEventCreate(&start_time);
    hipEventCreate(&stop);
    hipEventCreate(&epoch_start);
    hipEventCreate(&epoch_stop);

    float forward_time = 0, loss_time = 0, grad_time = 0, update_time = 0, total_time = 0;

    // Data parameters
    const float weight = 0.7f;
    const float bias = 0.3f;
    const float start = 0.0f;
    const float end = 10.0f;
    const float step = 0.000002f;

    const int n = static_cast<int>((end - start) / step);
    const int train_split = static_cast<int>(0.8 * n);
    const int test_split = n - train_split;

    // Print out the number of data points
    std::cout << "Total number of datapoints: " << n << std::endl;
    std::cout << "Number of training datapoints: " << train_split << std::endl;
    std::cout << "Number of testing datapoints: " << test_split << std::endl;

    // Create CUDA streams
    const int numberOfStreams = 4;
    hipStream_t streams[numberOfStreams];
    for (int i = 0; i < numberOfStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Allocate memory on the device
    float *d_X, *d_y, *d_w, *d_b, *d_y_pred, *d_loss, *d_w_grad, *d_b_grad;
    hiprandState *d_states;
    hipMalloc(&d_X, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_y_pred, n * sizeof(float));
    hipMalloc(&d_loss, sizeof(float));
    hipMalloc(&d_w, sizeof(float));
    hipMalloc(&d_b, sizeof(float));
    hipMalloc(&d_w_grad, sizeof(float));
    hipMalloc(&d_b_grad, sizeof(float));
    hipMalloc(&d_states, n * sizeof(hiprandState));

    hipEventRecord(start_time);

    // Initialize data on GPU
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    initialize_data<<<blocks, BLOCK_SIZE>>>(d_X, d_y, weight, bias, start, step, n, d_states);

    // Initialize weights
    float h_w = 0.83f;
    float h_b = 0.7645f;
    hipMemcpyAsync(d_w, &h_w, sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_b, &h_b, sizeof(float), hipMemcpyHostToDevice, streams[0]);

    // Training loop
    int epochs = 200;
    float lr = 0.01f;

    for (int epoch = 0; epoch < epochs; ++epoch) {
        hipEventRecord(epoch_start);

        // Zero the gradients and loss
        float zero = 0.0f;
        hipMemcpyAsync(d_loss, &zero, sizeof(float), hipMemcpyHostToDevice, streams[0]);
        hipMemcpyAsync(d_w_grad, &zero, sizeof(float), hipMemcpyHostToDevice, streams[1]);
        hipMemcpyAsync(d_b_grad, &zero, sizeof(float), hipMemcpyHostToDevice, streams[2]);

        // Forward pass
        hipEvent_t forward_start, forward_stop;
        hipEventCreate(&forward_start);
        hipEventCreate(&forward_stop);
        hipEventRecord(forward_start);

        forward_pass<<<blocks, BLOCK_SIZE>>>(d_X, d_w, d_b, d_y_pred, train_split);

        hipEventRecord(forward_stop);
        hipEventSynchronize(forward_stop);
        float forward_time_epoch;
        hipEventElapsedTime(&forward_time_epoch, forward_start, forward_stop);
        forward_time += forward_time_epoch;

        // Loss calculation
        hipEvent_t loss_start, loss_stop;
        hipEventCreate(&loss_start);
        hipEventCreate(&loss_stop);
        hipEventRecord(loss_start);

        calculate_loss<<<blocks, BLOCK_SIZE>>>(d_y_pred, d_y, d_loss, train_split);

        hipEventRecord(loss_stop);
        hipEventSynchronize(loss_stop);
        float loss_time_epoch;
        hipEventElapsedTime(&loss_time_epoch, loss_start, loss_stop);
        loss_time += loss_time_epoch;

        // Gradient calculation
        hipEvent_t grad_start, grad_stop;
        hipEventCreate(&grad_start);
        hipEventCreate(&grad_stop);
        hipEventRecord(grad_start);

        calculate_gradients<<<blocks, BLOCK_SIZE>>>(d_X, d_y_pred, d_y, d_w_grad, d_b_grad, train_split);

        hipEventRecord(grad_stop);
        hipEventSynchronize(grad_stop);
        float grad_time_epoch;
        hipEventElapsedTime(&grad_time_epoch, grad_start, grad_stop);
        grad_time += grad_time_epoch;

        // Update weights
        hipEvent_t update_start, update_stop;
        hipEventCreate(&update_start);
        hipEventCreate(&update_stop);
        hipEventRecord(update_start);

        update_weights<<<1, 1>>>(d_w, d_w_grad, d_b, d_b_grad, lr);

        hipEventRecord(update_stop);
        hipEventSynchronize(update_stop);
        float update_time_epoch;
        hipEventElapsedTime(&update_time_epoch, update_start, update_stop);
        update_time += update_time_epoch;

        hipEventRecord(epoch_stop);
        hipEventSynchronize(epoch_stop);
        float epoch_time;
        hipEventElapsedTime(&epoch_time, epoch_start, epoch_stop);
        total_time += epoch_time;

        // Print out what's happening every 10 epochs
        if (epoch % 10 == 0) {
            float h_loss, h_w, h_b;
            hipMemcpyAsync(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost, streams[0]);
            hipMemcpyAsync(&h_w, d_w, sizeof(float), hipMemcpyDeviceToHost, streams[1]);
            hipMemcpyAsync(&h_b, d_b, sizeof(float), hipMemcpyDeviceToHost, streams[2]);
            hipDeviceSynchronize();
            std::cout << "Epoch: " << epoch << " | Loss: " << h_loss << " | w: " << h_w << " | b: " << h_b 
                      << " | Epoch time: " << epoch_time << " ms" << std::endl;
        }

        hipEventDestroy(forward_start);
        hipEventDestroy(forward_stop);
        hipEventDestroy(loss_start);
        hipEventDestroy(loss_stop);
        hipEventDestroy(grad_start);
        hipEventDestroy(grad_stop);
        hipEventDestroy(update_start);
        hipEventDestroy(update_stop);
    }

    // Compute test loss
    float h_test_loss;
    hipMemsetAsync(d_loss, 0, sizeof(float), streams[0]);
    forward_pass<<<blocks, BLOCK_SIZE>>>(&d_X[train_split], d_w, d_b, &d_y_pred[train_split], test_split);
    calculate_loss<<<blocks, BLOCK_SIZE>>>(&d_y_pred[train_split], &d_y[train_split], d_loss, test_split);
    hipMemcpyAsync(&h_test_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost, streams[0]);

    // Copy final weights and bias back to host
    float h_w_final, h_b_final;
    hipMemcpyAsync(&h_w_final, d_w, sizeof(float), hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(&h_b_final, d_b, sizeof(float), hipMemcpyDeviceToHost, streams[1]);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_time, stop);

    std::cout << "Final weight: " << h_w_final << " (True: " << weight << ")" << std::endl;
    std::cout << "Final bias: " << h_b_final << " (True: " << bias << ")" << std::endl;
    std::cout << "Test loss: " << h_test_loss << std::endl;
    std::cout << "Training time: " << milliseconds << " ms" << std::endl;

    std::cout << "Average time per epoch:" << std::endl;
    std::cout << "Forward pass: " << forward_time / epochs << " ms" << std::endl;
    std::cout << "Loss calculation: " << loss_time / epochs << " ms" << std::endl;
    std::cout << "Gradient calculation: " << grad_time / epochs << " ms" << std::endl;
    std::cout << "Weight update: " << update_time / epochs << " ms" << std::endl;
    std::cout << "Total: " << total_time / epochs << " ms" << std::endl;

    // Free device memory
    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_y_pred);
    hipFree(d_loss);
    hipFree(d_w);
    hipFree(d_b);
    hipFree(d_w_grad);
    hipFree(d_b_grad);
    hipFree(d_states);

    // Destroy streams
    for (int i = 0; i < numberOfStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    // Destroy events
    hipEventDestroy(start_time);
    hipEventDestroy(stop);
    hipEventDestroy(epoch_start);
    hipEventDestroy(epoch_stop);

    return 0;
}

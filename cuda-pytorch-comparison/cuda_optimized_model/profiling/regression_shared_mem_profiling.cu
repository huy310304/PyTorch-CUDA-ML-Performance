#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 256

// CUDA kernel for data initialization
__global__ void initialize_data(float* X, float* y, float weight, float bias, float start, float step, int n, hiprandState* states) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        hiprand_init(1234, i, 0, &states[i]);
        X[i] = start + i * step;
        y[i] = weight * X[i] + bias + hiprand_normal(&states[i]) * 0.1f; // Add some noise
    }
}

// CUDA kernel for forward pass, loss calculation, and gradient calculation using shared memory
__global__ void fused_kernel(float* X, float* y_true, float* w, float* b, float* y_pred, float* loss, float* w_grad, float* b_grad, int n) {
    __shared__ float s_X[BLOCK_SIZE];
    __shared__ float s_y_true[BLOCK_SIZE];
    __shared__ float s_loss[BLOCK_SIZE];
    __shared__ float s_w_grad[BLOCK_SIZE];
    __shared__ float s_b_grad[BLOCK_SIZE];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    if (i < n) {
        s_X[tid] = X[i];
        s_y_true[tid] = y_true[i];
    }
    __syncthreads();

    float local_loss = 0.0f;
    float local_w_grad = 0.0f;
    float local_b_grad = 0.0f;

    if (i < n) {
        // Forward pass
        float pred = w[0] * s_X[tid] + b[0];
        y_pred[i] = pred;

        // Calculate the difference between prediction and true value
        float diff = pred - s_y_true[tid];

        // Compute local loss and gradients
        local_loss = diff * diff / (2 * n);  // Mean Squared Error
        local_w_grad = diff * s_X[tid] / n;
        local_b_grad = diff / n;
    }

    // Store local results in shared memory
    s_loss[tid] = local_loss;
    s_w_grad[tid] = local_w_grad;
    s_b_grad[tid] = local_b_grad;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            s_loss[tid] += s_loss[tid + stride];
            s_w_grad[tid] += s_w_grad[tid + stride];
            s_b_grad[tid] += s_b_grad[tid + stride];
        }
        __syncthreads();
    }

    // Write the block results to global memory
    if (tid == 0) {
        atomicAdd(loss, s_loss[0]);
        atomicAdd(w_grad, s_w_grad[0]);
        atomicAdd(b_grad, s_b_grad[0]);
    }
}

// CUDA kernel for updating weights
__global__ void update_weights(float* w, float* w_grad, float* b, float* b_grad, float lr) {
    w[0] -= lr * w_grad[0];
    b[0] -= lr * b_grad[0];
}

int main() {
    hipSetDevice(0);

    // Timing events
    hipEvent_t start_time, stop, epoch_start, epoch_stop;
    hipEventCreate(&start_time);
    hipEventCreate(&stop);
    hipEventCreate(&epoch_start);
    hipEventCreate(&epoch_stop);

    float forward_time = 0, loss_time = 0, grad_time = 0, update_time = 0, total_time = 0;

    // Data parameters
    const float weight = 0.7f;
    const float bias = 0.3f;
    const float start = 0.0f;
    const float end = 10.0f;
    const float step = 0.000002f;

    const int n = static_cast<int>((end - start) / step);
    const int train_split = static_cast<int>(0.8 * n);
    const int test_split = n - train_split;

    // Print out the number of data points
    std::cout << "Total number of datapoints: " << n << std::endl;
    std::cout << "Number of training datapoints: " << train_split << std::endl;
    std::cout << "Number of testing datapoints: " << test_split << std::endl;

    // Create CUDA streams
    const int numberOfStreams = 4;
    hipStream_t streams[numberOfStreams];
    for (int i = 0; i < numberOfStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Allocate memory on the device
    float *d_X, *d_y, *d_w, *d_b, *d_y_pred, *d_loss, *d_w_grad, *d_b_grad;
    hiprandState *d_states;
    hipMalloc(&d_X, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMalloc(&d_y_pred, n * sizeof(float));
    hipMalloc(&d_loss, sizeof(float));
    hipMalloc(&d_w, sizeof(float));
    hipMalloc(&d_b, sizeof(float));
    hipMalloc(&d_w_grad, sizeof(float));
    hipMalloc(&d_b_grad, sizeof(float));
    hipMalloc(&d_states, n * sizeof(hiprandState));

    hipEventRecord(start_time);

    // Initialize data on GPU
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    initialize_data<<<blocks, BLOCK_SIZE>>>(d_X, d_y, weight, bias, start, step, n, d_states);

    // Initialize weights
    float h_w = 0.83f;
    float h_b = 0.7645f;
    hipMemcpyAsync(d_w, &h_w, sizeof(float), hipMemcpyHostToDevice, streams[0]);
    hipMemcpyAsync(d_b, &h_b, sizeof(float), hipMemcpyHostToDevice, streams[0]);

    // Training loop
    int epochs = 200;
    float lr = 0.01f;

    for (int epoch = 0; epoch < epochs; ++epoch) {
        hipEventRecord(epoch_start);

        // Zero the gradients and loss
        float zero = 0.0f;
        hipMemcpyAsync(d_loss, &zero, sizeof(float), hipMemcpyHostToDevice, streams[0]);
        hipMemcpyAsync(d_w_grad, &zero, sizeof(float), hipMemcpyHostToDevice, streams[1]);
        hipMemcpyAsync(d_b_grad, &zero, sizeof(float), hipMemcpyHostToDevice, streams[2]);

        // Fused kernel for training data (using multiple streams)
        hipEvent_t fused_start, fused_stop;
        hipEventCreate(&fused_start);
        hipEventCreate(&fused_stop);
        hipEventRecord(fused_start);

        int segmentSize = train_split / numberOfStreams;
        for (int i = 0; i < numberOfStreams; ++i) {
            int offset = i * segmentSize;
            fused_kernel<<<(segmentSize + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, streams[i]>>>(
                &d_X[offset], &d_y[offset], d_w, d_b, 
                &d_y_pred[offset], d_loss, d_w_grad, d_b_grad, segmentSize);
        }

        hipEventRecord(fused_stop);
        hipEventSynchronize(fused_stop);
        float fused_time;
        hipEventElapsedTime(&fused_time, fused_start, fused_stop);
        forward_time += fused_time;
        loss_time += fused_time;
        grad_time += fused_time;

        // Update weights
        hipEvent_t update_start, update_stop;
        hipEventCreate(&update_start);
        hipEventCreate(&update_stop);
        hipEventRecord(update_start);

        update_weights<<<1, 1, 0, streams[0]>>>(d_w, d_w_grad, d_b, d_b_grad, lr);

        hipEventRecord(update_stop);
        hipEventSynchronize(update_stop);
        float update_time_epoch;
        hipEventElapsedTime(&update_time_epoch, update_start, update_stop);
        update_time += update_time_epoch;

        hipEventRecord(epoch_stop);
        hipEventSynchronize(epoch_stop);
        float epoch_time;
        hipEventElapsedTime(&epoch_time, epoch_start, epoch_stop);
        total_time += epoch_time;

        // Print out what's happening every 10 epochs
        if (epoch % 10 == 0) {
            float h_loss, h_w, h_b;
            hipMemcpyAsync(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost, streams[0]);
            hipMemcpyAsync(&h_w, d_w, sizeof(float), hipMemcpyDeviceToHost, streams[1]);
            hipMemcpyAsync(&h_b, d_b, sizeof(float), hipMemcpyDeviceToHost, streams[2]);
            hipDeviceSynchronize();
            std::cout << "Epoch: " << epoch << " | Loss: " << h_loss << " | w: " << h_w << " | b: " << h_b 
                      << " | Epoch time: " << epoch_time << " ms" << std::endl;
        }

        hipEventDestroy(fused_start);
        hipEventDestroy(fused_stop);
        hipEventDestroy(update_start);
        hipEventDestroy(update_stop);
    }

    // Compute test loss
    float h_test_loss;
    hipMemsetAsync(d_loss, 0, sizeof(float), streams[0]);
    fused_kernel<<<(test_split + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, streams[0]>>>(
        &d_X[train_split], &d_y[train_split], d_w, d_b, &d_y_pred[train_split], d_loss, d_w_grad, d_b_grad, test_split);
    hipMemcpyAsync(&h_test_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost, streams[0]);

    // Copy final weights and bias back to host
    float h_w_final, h_b_final;
    hipMemcpyAsync(&h_w_final, d_w, sizeof(float), hipMemcpyDeviceToHost, streams[0]);
    hipMemcpyAsync(&h_b_final, d_b, sizeof(float), hipMemcpyDeviceToHost, streams[1]);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_time, stop);

    std::cout << "Final weight: " << h_w_final << " (True: " << weight << ")" << std::endl;
    std::cout << "Final bias: " << h_b_final << " (True: " << bias << ")" << std::endl;
    std::cout << "Test loss: " << h_test_loss << std::endl;
    std::cout << "Training time: " << milliseconds << " ms" << std::endl;

    std::cout << "Average time per epoch:" << std::endl;
    std::cout << "Forward pass + Loss calculation + Gradient calculation: " << forward_time / epochs << " ms" << std::endl;
    std::cout << "Weight update: " << update_time / epochs << " ms" << std::endl;
    std::cout << "Total: " << total_time / epochs << " ms" << std::endl;

    // Free device memory
    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_y_pred);
    hipFree(d_loss);
    hipFree(d_w);
    hipFree(d_b);
    hipFree(d_w_grad);
    hipFree(d_b_grad);
    hipFree(d_states);

    // Destroy streams
    for (int i = 0; i < numberOfStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    // Destroy events
    hipEventDestroy(start_time);
    hipEventDestroy(stop);
    hipEventDestroy(epoch_start);
    hipEventDestroy(epoch_stop);

    return 0;
}
